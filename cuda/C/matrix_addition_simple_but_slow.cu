
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <inttypes.h>
#include <time.h>
#define SIZE 102400

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;

  //for (i = 0; i < n; ++i)
  if (i < n)
    c[i] = a[i] + b[i];
}

int main()
{ 
  int *a, *b, *c;
  int blockSize, gridSize;

  hipMallocManaged(&a, SIZE * sizeof(int));
  hipMallocManaged(&b, SIZE * sizeof(int));
  hipMallocManaged(&c, SIZE * sizeof(int));

  for (int i = 0; i < SIZE; ++i)
  {
    a[i] = i;
    b[i] = i;
    c[i] = 0;
  }

  blockSize = 1024;
  gridSize = (int)ceil((float)SIZE/blockSize);

  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);
  
  VectorAdd <<<gridSize, blockSize>>> (a, b, c, SIZE);
  
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  uint64_t delta_us = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
    
  hipDeviceSynchronize();

  for (int i = 0; i < SIZE; ++i)
    printf("c[%d] = %d\n", i, c[i]);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  
  printf("Code executed in ""%"PRId64" milliseconds.\n", delta_us);
  
  return 0;
}
