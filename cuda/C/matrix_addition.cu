
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <time.h>
#define SIZE 102400

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
  // Get our global thread ID
  int i = blockIdx.x*blockDim.x+threadIdx.x;

  //for (i = 0; i < n; ++i) // replaced

  // Make sure we do not go out of bounds
  if (i < n)
    c[i] = a[i] + b[i];
}

int main( int argc, char* argv[] )
{
  int *cpu_a, *cpu_b, *cpu_c;
  int *gpu_a, *gpu_b, *gpu_c;
  int blockSize, gridSize;
  

  size_t bytes = SIZE * sizeof(int);

  // Allocate memory for each vector on CPU
  cpu_a = (int *)malloc(bytes);
  cpu_b = (int *)malloc(bytes);
  cpu_c = (int *)malloc(bytes);
  
  // Allocate memory for each vector on GPU
  hipMalloc(&gpu_a, bytes);
  hipMalloc(&gpu_b, bytes);
  hipMalloc(&gpu_c, bytes);

  for (int i = 0; i < SIZE; ++i)
  {
    cpu_a[i] = i;
    cpu_b[i] = i;
    cpu_c[i] = 0;
  }
  
  // Copy host vectors to device
  hipMemcpy( gpu_a, cpu_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy( gpu_b, cpu_b, bytes, hipMemcpyHostToDevice);
  
  // Number of threads in each thread block
  blockSize = 1024;
 
  // Number of thread blocks in grid
  gridSize = (int)ceil((float)SIZE/blockSize);

  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);  

  VectorAdd <<<gridSize, blockSize>>> (gpu_a, gpu_b, gpu_c, SIZE);

  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  uint64_t delta_us = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
  
  hipMemcpy(cpu_c, gpu_c, bytes, hipMemcpyDeviceToHost);

  for (int i = 0; i < SIZE; ++i)
    printf("c[%d] = %d\n", i, cpu_c[i]);

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_c);

  free(cpu_a);
  free(cpu_b);
  free(cpu_c);

  

  
  printf("Vector addition executed in ""%"PRId64" milliseconds.\n", delta_us);
  
  return 0;
}
