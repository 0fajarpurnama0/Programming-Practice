
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <inttypes.h>
#include <time.h>
#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
  int i = blockIdx.x*blockDim.x+threadIdx.x;

  //for (i = 0; i < n; ++i)
  if (i < n)
    c[i] = a[i] + b[i];
}

int main()
{
  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);
  
  int *a, *b, *c;

  hipMallocManaged(&a, SIZE * sizeof(int));
  hipMallocManaged(&b, SIZE * sizeof(int));
  hipMallocManaged(&c, SIZE * sizeof(int));

  for (int i = 0; i < SIZE; ++i)
  {
    a[i] = i;
    b[i] = i;
    c[i] = 0;
  }

  VectorAdd <<<1, SIZE>>> (a, b, c, SIZE);
  
  hipDeviceSynchronize();

  for (int i = 0; i < SIZE; ++i)
    printf("c[%d] = %d\n", i, c[i]);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  uint64_t delta_us = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
  
  printf("Code executed in ""%"PRId64" milliseconds.\n", delta_us);
  
  return 0;
}
