
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <ctime>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__ void add(int n, float *x, float *y)
{

  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i+= stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

  float *x, *y;
  
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Timer Start
  std::clock_t start;
  double duration;
  start = std::clock();
  
  int blockSize = 1024;
  int numBlocks = (N + blockSize -1) / blockSize;
  // Run kernel on 1M elements on the CPU
  add<<<numBlocks, blockSize>>>(N, x, y);

  duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
  // Timer Stop
   
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
  {
    maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "y[" << i << "] = " << y[i] << std::endl;
  }
  std::cout << "Max error: " << maxError << std::endl;
  std::cout<<"Duration for matrix addition: "<< duration <<'\n';

  // Free memory

  hipFree(x);
  hipFree(y);

  //delete [] x;
  //delete [] y;

  return 0;
}
