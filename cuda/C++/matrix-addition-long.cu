
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <ctime>

// CUDA Kernel function to add the elements of two arrays on the GPU

__global__ void add(int n, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 50000; // 50k elements

  float *cpu_x, *cpu_y, *gpu_x, *gpu_y;

  size_t bytes = N * sizeof(float);
  
  cpu_x = (float *)malloc(bytes);
  cpu_y = (float *)malloc(bytes);

  hipMalloc(&gpu_x, bytes);
  hipMalloc(&gpu_y, bytes);
  
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    cpu_x[i] = 1.0f;
    cpu_y[i] = 2.0f;
  }
  
  hipMemcpy( gpu_x, cpu_x, bytes, hipMemcpyHostToDevice);
  hipMemcpy( gpu_y, cpu_y, bytes, hipMemcpyHostToDevice);
  
  int blockSize = 1024;
  int numBlocks = (N + blockSize -1) / blockSize;
  
  // Timer Start
  std::clock_t start;
  double duration;
  start = std::clock();
  
  // Run kernel on 1M elements on the GPU
  add<<<numBlocks, blockSize>>>(N, gpu_x, gpu_y);

  duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
  // Timer Stop
   
  hipMemcpy(cpu_y, gpu_y, bytes, hipMemcpyDeviceToHost);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
  {
    maxError = fmax(maxError, fabs(cpu_y[i] - 3.0f));
    std::cout << "y[" << i << "] = " << cpu_y[i] << std::endl;
  }
  std::cout << "Max error: " << maxError << std::endl;
  std::cout<<"Duration for matrix addition: "<< duration <<'\n';

  // Free memory

  hipFree(gpu_x);
  hipFree(gpu_y);

  delete [] cpu_x;
  delete [] cpu_y;

  return 0;
}
